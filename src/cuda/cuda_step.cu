#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <vector>
#include <fstream>
#include <math.h>

#include </usr/local/cuda/include/hip/hip_runtime.h>
#include "cuda_step.h"

using namespace std;

#define CUDA_BLOCK 32

__global__ void calculateStepParticles(double *forceXd, double *forceYd, double *forceZd, double *posXd, double *posYd, double *posZd,
				       double *massd, double *velXd, double *velYd, double *velZd, double h, int sizeX, int numParticlesPadded, double e2) {
   int tNdx = blockIdx.x * CUDA_BLOCK + threadIdx.x;

   double forcex = 0, forcey = 0, forcez = 0;
   for(int i = 0; i < numParticlesPadded; i++) {
      if(tNdx != i) {
	      double rijx = posXd[i] - posXd[tNdx];
	      double rijy = posYd[i] - posYd[tNdx];
	      double rijz = posZd[i] - posZd[tNdx];
	      double normalize = pow(rijx*rijx + rijy*rijy + rijz*rijz, 0.5);
	      double rsquared = normalize * normalize;
	      double numerator = massd[tNdx] * massd[i];
	      double denominator = pow(rsquared + e2, 3.0/2);
	      double multiplier = numerator/denominator;
    
	      forcex += multiplier * rijx;
	      forcey += multiplier * rijy;
	      forcez += multiplier * rijz;
      }
   }
   

   forceXd[tNdx] = forcex;
   forceYd[tNdx] = forcey;
   forceZd[tNdx] = forcez;

   velXd[tNdx] += (h * 1.0/massd[tNdx]) * forceXd[tNdx];
   velYd[tNdx] += (h * 1.0/massd[tNdx]) * forceYd[tNdx];
   velZd[tNdx] += (h * 1.0/massd[tNdx]) * forceZd[tNdx];

   posXd[tNdx] += (h * velXd[tNdx]);
   posYd[tNdx] += (h * velYd[tNdx]);
   posZd[tNdx] += (h * velZd[tNdx]);
}

void stepParticles(vector<double> &positionx, vector<double> &positiony , vector<double> &positionz, vector<double> &masses,
		   vector<double> &velocityx, vector<double> &velocityy, vector<double> &velocityz, double h, double *t, double e2)
{
    int sizeX = positionx.size();
    int sizeY = positiony.size();
    int sizeZ = positionz.size();

    int numParticlesPadded = (int) (ceil(1.0 * sizeX / CUDA_BLOCK)) * CUDA_BLOCK;

    // setting up device variables
    double *forceXd = NULL, *forceYd = NULL, *forceZd = NULL;
    double *posXd = NULL, *posYd = NULL, *posZd = NULL;
    double *massd = NULL;
    double *velXd = NULL, *velYd = NULL, *velZd = NULL;

    // hipMalloc with padding
    hipError_t cudaerror = hipMalloc((void **) &forceXd, sizeof(double) * numParticlesPadded);
    hipMalloc((void **) &forceYd, sizeof(double) * numParticlesPadded);
    hipMalloc((void **) &forceZd, sizeof(double) * numParticlesPadded);
    cudaerror = hipMalloc((void **) &posXd, sizeof(double) * numParticlesPadded);
    hipMalloc((void **) &posYd, sizeof(double) * numParticlesPadded);
    hipMalloc((void **) &posZd, sizeof(double) * numParticlesPadded);
    hipMalloc((void **) &massd, sizeof(double) * numParticlesPadded);
    hipMalloc((void **) &velXd, sizeof(double) * numParticlesPadded);
    hipMalloc((void **) &velYd, sizeof(double) * numParticlesPadded);
    cudaerror = hipMalloc((void **) &velZd, sizeof(double) * numParticlesPadded);

    // copy data from host to device in preparation for calculation
    cudaerror = hipMemcpy(posXd, &positionx[0], sizeof(double) * sizeX, hipMemcpyHostToDevice);
    hipMemcpy(posYd, &positiony[0], sizeof(double) * sizeY, hipMemcpyHostToDevice);
    hipMemcpy(posZd, &positionz[0], sizeof(double) * sizeZ, hipMemcpyHostToDevice);
    hipMemcpy(massd, &masses[0], sizeof(double) * sizeX, hipMemcpyHostToDevice);
    hipMemcpy(velXd, &velocityx[0], sizeof(double) * sizeX, hipMemcpyHostToDevice);
    hipMemcpy(velYd, &velocityy[0], sizeof(double) * sizeY, hipMemcpyHostToDevice);
    hipMemcpy(velZd, &velocityz[0], sizeof(double) * sizeZ, hipMemcpyHostToDevice);

    dim3 dimBlock(CUDA_BLOCK, 1);
    dim3 dimGrid(numParticlesPadded / CUDA_BLOCK, 1);

    // kernel call
    calculateStepParticles<<<dimGrid, dimBlock>>>(forceXd, forceYd, forceZd, posXd, posYd, posZd, massd, velXd, velYd, velZd, h, sizeX, numParticlesPadded, e2);

    // copy data back from device to host
    hipMemcpy(&positionx[0], posXd, sizeof(double) * sizeX, hipMemcpyDeviceToHost);
    hipMemcpy(&positiony[0], posYd, sizeof(double) * sizeY, hipMemcpyDeviceToHost);
    hipMemcpy(&positionz[0], posZd, sizeof(double) * sizeZ, hipMemcpyDeviceToHost);
    hipMemcpy(&velocityx[0], velXd, sizeof(double) * sizeX, hipMemcpyDeviceToHost);
    hipMemcpy(&velocityy[0], velYd, sizeof(double) * sizeY, hipMemcpyDeviceToHost);
    hipMemcpy(&velocityz[0], velZd, sizeof(double) * sizeZ, hipMemcpyDeviceToHost);

    // free hipMalloc
    hipFree(forceXd);
    hipFree(forceYd);
    hipFree(forceZd);
    hipFree(posXd);
    hipFree(posYd);
    hipFree(posZd);
    hipFree(massd);
    hipFree(velXd);
    hipFree(velYd);
    hipFree(velZd);

    *t += h;
}
